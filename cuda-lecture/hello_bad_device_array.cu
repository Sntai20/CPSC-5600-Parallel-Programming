
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>

void auto_throw(hipError_t status) {
    if(status != hipSuccess) {
        std::string message = "ERROR: '";
        message += hipGetErrorString(status);
        message +="'\n";
        throw std::runtime_error(message);
    }
}

__global__ void array_square(int* array, size_t size) {
    for(size_t i=threadIdx.x; i<size; i+=32){
        array[i] = array[i] * array[i];
    }
}

void print_array(int* array, size_t size) {
    for(size_t i=0; i<size; i++){
        if(i != 0){
            std::cout << ',';
        }
        std::cout << array[i];
    }
    std::cout << '\n';
}

int main(int argc, char *argv[]) {
    size_t size = (argc>1) ? atoi(argv[1]) : 0;

    int *array = new int[size];
    hipMallocManaged(
        &array,
        size*sizeof(int));

    for(size_t i=0; i<size; i++){
        array[i] = i;
    }

    int *device_array;
    // cudaMalloc(&device_array, size*sizeof(int));

    // Copy the array to the device.
    // cudaMemcpy(
    //     device_array,
    //     array,
    //     size*sizeof(int),
    //     cudaMemcpyHostToDevice);

    print_array(array,size);

    // array_square<<<1,32>>>(array,size);
    array_square<<<1,32>>>(array,size);
    auto_throw(hipDeviceSynchronize());

    // Copy the array back to the host.
    // cudaMemcpy(
    //     array,
    //     device_array,
    //     size*sizeof(int),
    //     cudaMemcpyDeviceToHost);

    print_array(array,size);

    // delete[] array;
    hipFree(device_array);
    return 0;
}